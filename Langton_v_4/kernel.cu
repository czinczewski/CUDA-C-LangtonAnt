#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void searchKrenel(int *dev_width, int *dev_Ants_Inf, int *dev_Ants_Size, int *dev_Ant_Array, int *dev_Ant_Map){
	int i = threadIdx.x;
	int x = dev_Ant_Array[*dev_Ants_Inf * i];
	int y = dev_Ant_Array[*dev_Ants_Inf * i + 1];

	hiprandState_t state;
	hiprand_init(0, i, 0, &state);

	for (int search = 0; search < *dev_Ants_Size; search++){
		if (search != i && x >= (dev_Ant_Array[search * *dev_Ants_Inf] - 2) && x <= dev_Ant_Array[search * *dev_Ants_Inf] 
			&& y / (*dev_width) <= dev_Ant_Array[search* *dev_Ants_Inf + 1] / (*dev_width) + 2 && y >= dev_Ant_Array[search* *dev_Ants_Inf + 1]){
			printf("\n Mrowka %i znalazla %i, x:%i y:%i k:%i", i, search, dev_Ant_Array[i], dev_Ant_Array[i + 1] / (*dev_width), dev_Ant_Array[i + 2]);
			// czyszcenie poprzedniej pozycji
				dev_Ant_Map[y + x] = 0;
			// pozycja x
				dev_Ant_Array[*dev_Ants_Inf * i] = 1 + (hiprand(&state) % (*dev_width - 2));
			// pozycja y
				dev_Ant_Array[*dev_Ants_Inf * i + 1] = *dev_width * (1 + (hiprand(&state) % (*dev_width - 2)));
			// kierunek
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 1 + (hiprand(&state) % 4);
			//wypisanie
				printf(" x2:%i y2:%i k2:%i", dev_Ant_Array[i], dev_Ant_Array[i + 1] / (*dev_width), dev_Ant_Array[i + 2]);
				dev_Ant_Map[dev_Ant_Array[*dev_Ants_Inf *i + 1] + dev_Ant_Array[*dev_Ants_Inf *i]] = dev_Ant_Array[*dev_Ants_Inf *i + 2];
		}
		__syncthreads();	//synchronizacja w�tk�w
	}
}
__global__ void findFailsKernel(int *dev_width, int *dev_Ants_Inf, int *dev_Ants_Size, int *dev_Ant_Array, int *dev_Ant_Map){
	int i = threadIdx.x;

	int x = dev_Ant_Array[*dev_Ants_Inf * i];
	int y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
	int k = dev_Ant_Array[*dev_Ants_Inf * i + 2];

	hiprandState_t state;
	hiprand_init(1, i, 0, &state);
	if (x >= *dev_width || x < 0|| y < 0 || y >= *dev_width * *dev_width || k > 4 || k < 0){
		printf("\nPoprawiam mrowke %i\n", i);
		// czyszcenie nieprawid�owej pozycji
			dev_Ant_Map[y + x] = 0;
		// usuwanie b��d�w
			if (x >= *dev_width || x < 0){
				dev_Ant_Array[*dev_Ants_Inf * i] = 1 + (hiprand(&state) % (*dev_width - 2));
				x = dev_Ant_Array[*dev_Ants_Inf * i];
			}
			__syncthreads();	//synchronizacja w�tk�w
			if (y >= *dev_width * *dev_width || y < 0){
				dev_Ant_Array[*dev_Ants_Inf * i + 1] = *dev_width * (1 + (hiprand(&state) % (*dev_width - 2)));
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
			}
			__syncthreads();	//synchronizacja w�tk�w
			if (k > 4 || k < 0){
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 1 + (hiprand(&state) % 4);
				k = dev_Ant_Array[*dev_Ants_Inf * i + 2];
			}
			__syncthreads();	//synchronizacja w�tk�w
		// zapisanie do nowej prawid�owej pozycji
			dev_Ant_Map[dev_Ant_Array[*dev_Ants_Inf * i + 1] + dev_Ant_Array[*dev_Ants_Inf * i]] = dev_Ant_Array[*dev_Ants_Inf * i + 2];
	}
	__syncthreads();	//synchronizacja w�tk�w		
}
__global__ void alghoritmKernel(int *dev_width, int *dev_Ants_Inf, int *dev_Ant_Array, int *dev_Ant_Map, int *dev_Col_Map){
	int i = threadIdx.x;
	//for (int steps = 0; steps < 100; steps++){ //obliczanie danej liczby krok�w na raz przed przes��niem na cpu

		int x = dev_Ant_Array[*dev_Ants_Inf * i];
		int y = dev_Ant_Array[*dev_Ants_Inf * i + 1];

		if (dev_Col_Map[y + x] == 0){
			dev_Col_Map[y + x] = 1;

			if (dev_Ant_Map[y + x] == 1){ // obr�cona do g�ry
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 2; // zmian kierunku
				dev_Ant_Map[y + x] = 0;

				if ( x == *dev_width - 1){
					dev_Ant_Array[*dev_Ants_Inf * i] = 0;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1]; // zmian poz y
				}else{
					dev_Ant_Array[*dev_Ants_Inf * i]		= dev_Ant_Array[*dev_Ants_Inf * i] + 1;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1]	= dev_Ant_Array[*dev_Ants_Inf * i + 1]; // zmian poz y
				}

				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}else if (dev_Ant_Map[y + x] == 2){ // obr�cona w prawo
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 3; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (y/(*dev_width) == *dev_width - 1){
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = 0; // zmian poz y
				}
				else{
					dev_Ant_Array[*dev_Ants_Inf * i]		= dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1]	= ((dev_Ant_Array[*dev_Ants_Inf * i + 1]/ *dev_width) + 1)*(*dev_width) ;// zmian poz y
				}

				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}else if (dev_Ant_Map[y + x] == 3){ // obr�cona w d�
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 4; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (x == 0){
					dev_Ant_Array[*dev_Ants_Inf * i] = *dev_width-1;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1];// zmian poz y
				}
				else{
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i] - 1;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1];// zmian poz y
				}

				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}else if (dev_Ant_Map[y + x] == 4){ // obr�cona w lewo
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 1; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (y/(*dev_width) == 0){
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = (*dev_width - 1)*(*dev_width);// zmian poz y
				}
				else{
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = ((dev_Ant_Array[*dev_Ants_Inf * i + 1] / *dev_width) - 1)*(*dev_width);// zmian poz y
				}

				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}
		}else if (dev_Col_Map[y + x] == 1){
			dev_Col_Map[y + x] = 0;
		
			if (dev_Ant_Map[y + x] == 1){ // obr�cona do g�ry
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 4; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (x == 0){
					dev_Ant_Array[*dev_Ants_Inf * i] = *dev_width - 1 ;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1];// zmian poz y
				}else{
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i] - 1;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1];// zmian poz y
				}
			
				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}else if (dev_Ant_Map[y + x] == 2){ // obr�cona w prawo
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 1; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (y == 0){
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = ((*dev_width) - 1)*(*dev_width);// zmian poz y
				}else {
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = ((dev_Ant_Array[*dev_Ants_Inf * i + 1] / *dev_width) - 1)*(*dev_width);// zmian poz y

				}
			
				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}else if (dev_Ant_Map[y + x] == 3){ // obr�cona w d�
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 2; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (x == *dev_width - 1){
					dev_Ant_Array[*dev_Ants_Inf * i] = 0;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1];// zmian poz y
				}else{
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i] + 1;	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = dev_Ant_Array[*dev_Ants_Inf * i + 1];// zmian poz y
				}

				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}else if (dev_Ant_Map[y + x] == 4){ // obr�cona w lewo
				dev_Ant_Array[*dev_Ants_Inf * i + 2] = 3; // zmian kierunku
				dev_Ant_Map[y + x] = 0;
				if (y/(*dev_width) == *dev_width - 1){
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = 0;// zmian poz y
				}else{
					dev_Ant_Array[*dev_Ants_Inf * i] = dev_Ant_Array[*dev_Ants_Inf * i];	// zmian poz x
					dev_Ant_Array[*dev_Ants_Inf * i + 1] = ((dev_Ant_Array[*dev_Ants_Inf * i + 1] / *dev_width) + 1)*(*dev_width);// zmian poz y
				}
			
				x = dev_Ant_Array[*dev_Ants_Inf * i];
				y = dev_Ant_Array[*dev_Ants_Inf * i + 1];
				dev_Ant_Map[y + x] = dev_Ant_Array[*dev_Ants_Inf * i + 2];

			}
		}
		// synchronizacja
		__syncthreads();
	//}
}

void clear_Col_Map(int width, int Col_Map[64 * 64]){
	for (int i = 0; i < width*width; i++){
		if (i < width || i%width == 0 || i%width == width - 1 || i > width*(width - 1)){
			//Col_Map[i] = 1;	// tworze ramk� nie do przej�cia
			Col_Map[i] = 0;
		}
		else{
			Col_Map[i] = 0;
		}
	}
};
void clear_Map(int width, int Map[64 * 64]){
	for (int i = 0; i < width*width; i++){
		Map[i] = 0;
	}
};
void createAnts(int width, int Ants_Inf, int Ants_Size, int Ant_Array[100 * 3], int Ant_Map[64 * 64]){
	srand(time(NULL));
	for (int i = 0; i < Ants_Inf*Ants_Size; i += Ants_Inf){
		// pozycja x
		Ant_Array[i] = 1 + rand() % (width - 2);
		// pozycja y
		Ant_Array[i + 1] = width * (1 + rand() % (width - 2));
		// kierunek
		Ant_Array[i + 2] = 1 + rand() % 4;
		//wypisanie
		Ant_Map[Ant_Array[i + 1] + Ant_Array[i]] = Ant_Array[i + 2];
	}
}
void show_Col_Map(int width, int Col_Map[64 * 64], int Ant_Map[64 * 64]){
	printf("\n  ");
	for (int i = 0; i < width*width; i++){
		// wpisywanie numeru wiersza
		if (i%width == 0 && i < width * 10){
			printf("\n   %i ", i / width);
		}
		else if (i%width == 0 && i >= width * 10){
			printf("\n  %i ", i / width);
		}
		// wypisywanie komorek mapy
		if (Ant_Map[i] == 0){
			if (Col_Map[i] == 0){
				printf(" ");
			}
			else{
				printf("#");
			}
		}
		else{
			if (Ant_Map[i] == 1){
				printf("^");
			}
			if (Ant_Map[i] == 2){
				printf(">");
			}
			if (Ant_Map[i] == 3){
				printf("#");
			}
			if (Ant_Map[i] == 4){
				printf("<");
			}
		}

	}
}
void show_2_Map(int width, int Col_Map[64 * 64], int Ant_Map[64 * 64]){
	printf("\n\t Mapa Zycia %i x %i \t\t\t\t\t\t\t\t Mapa Mrowek %i x %i", width, width, width, width);
	printf("\n"); printf("\n     ");
	for (int k = 0; k < width; k++){
		printf(" _");
	}
	printf("\t\t     ");
	for (int k = 0; k < width; k++){
		printf(" _");
	}

	for (int i = 0; i < width*width; i++){
		// wpisywanie numeru wiersza
		if (i%width == 0 && i < width * 10){
			printf("|\n   %i ", i / width);
		}
		else if (i%width == 0 && i >= width * 10){
			printf("|\n  %i ", i / width);
		}
		// wypisywanie komorek mapy
		if (Col_Map[i] == 0){
			printf("|_");
		}
		else{
			printf("|#");
		}
		// wypisywanie mapy mrowek
		if (i%width == width - 1){
			printf("|\t\t");
			if (i%width == (width - 1) && i < width * 10){
				printf("   %i ", (i - width + 1) / width);
			}
			else if (i%width == (width - 1) && i >= width * 10){
				printf("  %i ", (i - width + 1) / width);
			}
			for (int k = 0; k < width; k++){
				if (Ant_Map[(i - width + 1) + k] == 0){
					printf("|_");
				}
				else{
					if (Ant_Map[(i - width + 1) + k] == 1){
						printf("|^");
					}
					if (Ant_Map[(i - width + 1) + k] == 2){
						printf("|>");
					}
					if (Ant_Map[(i - width + 1) + k] == 3){
						printf("|#");
					}
					if (Ant_Map[(i - width + 1) + k] == 4){
						printf("|<");
					}
				}
			}
		}

	}
}
void show_Ants(int width, int Ants_Inf, int Ants_Size, int Ant_Array[100 * 3]){
	printf("\n\n\n\t | Ant\t| x\t| y\t| kier\t|\n");
	printf("\t --------------------------------\n \t ");
	for (int i = 0; i < Ants_Size*Ants_Inf; i += Ants_Inf){
		printf("| %i\t| %i\t| %i\t| %i\t", i / Ants_Inf, Ant_Array[i], Ant_Array[i + 1] / width, Ant_Array[i + 2]);
		printf("|\n\t --------------------------------\n\t ");
	}
}

int main()
{
	const int Ants_Size = 16; // ilo�� mr�wek
	const int Ants_Inf = 3; // ilo�� danych o mr�wce
	const int width = 64;

	int Col_Map[width*width];
	int Ant_Map[width*width];
	int Ant_Array[Ants_Size*Ants_Inf];

	int *dev_Ants_Size; // ilo�� mr�wek
	int *dev_Ants_Inf; // ilo�� danych o mr�wce
	int *dev_width;

	int *dev_Col_Map;
	int *dev_Ant_Map;
	int *dev_Ant_Array;

//Alokowanie pami�ci na GPU
	hipMalloc((void**)&dev_Ants_Size, sizeof(int));
	hipMalloc((void**)&dev_Ants_Inf, sizeof(int));
	hipMalloc((void**)&dev_width, sizeof(int));

	hipMalloc((void**)&dev_Col_Map, width*width * sizeof(int));
	hipMalloc((void**)&dev_Ant_Map, width*width * sizeof(int));
	hipMalloc((void**)&dev_Ant_Array, Ants_Size * Ants_Inf * sizeof(int));

//Tworzenie danych poczatkowych
	clear_Col_Map(width, Col_Map);
	clear_Map(width, Ant_Map);
	createAnts(width, Ants_Inf, Ants_Size, Ant_Array, Ant_Map);

//Kopiowanie danych z CPU na GPU
	hipMemcpy(dev_Ants_Size, &Ants_Size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ants_Inf, &Ants_Inf, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_width, &width, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dev_Col_Map, Col_Map, width * width * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ant_Map, Ant_Map, width * width * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ant_Array, Ant_Array, Ants_Size * Ants_Inf * sizeof(int), hipMemcpyHostToDevice);
	
	int step = 0;
// Niesko�czoncza p�tla do algorytmu
	while (step < 1000){
		//	system("cls"); // czyszczenie konsoli
			printf("\n\n  Langthon Ant's Algorithm \tStep: %i\n", step);
		// Poszukiwnie b��d�w
			findFailsKernel << < 1, Ants_Size >> >(dev_width, dev_Ants_Inf, dev_Ants_Size, dev_Ant_Array, dev_Ant_Map);
		// Przeszukanie
			searchKrenel	<<< 1, Ants_Size >>>(dev_width, dev_Ants_Inf, dev_Ants_Size, dev_Ant_Array, dev_Ant_Map); //wyszykiwanie zbli�e�
		// Poszukiwnie b��d�w
			findFailsKernel << < 1, Ants_Size >> >(dev_width, dev_Ants_Inf, dev_Ants_Size, dev_Ant_Array, dev_Ant_Map);
		// Algorytm
			alghoritmKernel <<< 1, Ants_Size >>>(dev_width, dev_Ants_Inf, dev_Ant_Array, dev_Ant_Map, dev_Col_Map); //poruszenie mr�wek
		//Kopiowanie danych z GPU na CPU
			hipMemcpy(Col_Map, dev_Col_Map, width * width * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(Ant_Map, dev_Ant_Map, width * width * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(Ant_Array, dev_Ant_Array, Ants_Size * Ants_Inf * sizeof(int), hipMemcpyDeviceToHost);
		//wyrysowanie
			//show_Col_Map(width, Col_Map, Ant_Map);			// wyrysowanie jednej mapy zawierajacej informacje o kom�rkach i pozycji mr�wek
			show_2_Map(width, Col_Map, Ant_Map);				// wyrysowanie mapy �ywych/martwych kom�rek i mapy istniejacych mr�wek
			show_Ants(width, Ants_Inf, Ants_Size, Ant_Array);	// wypianie danych o mr�wkach

		step++;
	//getchar();  // pauza po kroku
	}

	getchar(); getchar();

	hipFree(dev_Col_Map);
	hipFree(dev_Ant_Map);
	hipFree(dev_Ant_Array);
	return 0;
}
